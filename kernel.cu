#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <raylib.h>
#include <cstdlib>
#include <ctime>
#include <cmath>

// Constants
#define MAX_PARTICLES 5000
#define BLOCK_SIZE 256

/////// CPU //////////////////////
// 2000 = 14 FPS                //
// 1000 = 53 FPS                //
// 500 = 142 FPS                //
//////////////////////////////////
/////// GPU first version ////////
// 5000 = 40 FPS                //
// 4000 = 50 FPS                //
// 3000 = 66 FPS                //
// 2000 = 96 FPS                //
// 1000 = 142 FPS               //
// 500 = 144 FPS                //
//////////////////////////////////
// 
__constant__ float d_MAX_DISTANCE;
__constant__ float d_MIN_DISTANCE;
__constant__ float d_FORCE_STRENGTH;
__constant__ float d_MIN_COLLISION_DISTANCE;
__constant__ float d_MAX_SPEED;
__constant__ float d_MIN_SPEED;

// Particle struct definition
struct Particle {
    Vector2 position;
    Vector2 velocity;
    Color color;
};

// Cap speed of a particle
__device__ void CapSpeed(Vector2& velocity, float maxSpeed, float minSpeed) {
    float speed = sqrtf(velocity.x * velocity.x + velocity.y * velocity.y);
    if (speed > maxSpeed) {
        velocity.x = (velocity.x / speed) * maxSpeed;
        velocity.y = (velocity.y / speed) * maxSpeed;
    }
    if (speed < minSpeed && speed > 0) {
        velocity.x = (velocity.x / speed) * minSpeed;
        velocity.y = (velocity.y / speed) * minSpeed;
    }
}

// Kernel to update particle interactions
__global__ void UpdateParticleInteractions(Particle* particles, int particleCount, int screenWidth, int screenHeight) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleCount) return;

    Particle& p1 = particles[i];
    for (int j = 0; j < particleCount; ++j) {
        if (i == j) continue;

        Particle& p2 = particles[j];
        float dx = p2.position.x - p1.position.x;
        float dy = p2.position.y - p1.position.y;
        float distance = sqrtf(dx * dx + dy * dy);

        if (distance < d_MAX_DISTANCE && distance > d_MIN_DISTANCE) {
            float force = -d_FORCE_STRENGTH / distance;
            Vector2 direction = { dx / distance, dy / distance };
            p1.velocity.x += direction.x * force;
            p1.velocity.y += direction.y * force;
        }

        if (distance < d_MIN_COLLISION_DISTANCE) {
            Vector2 collisionDirection = { dx / distance, dy / distance };
            p1.velocity.x -= collisionDirection.x * d_FORCE_STRENGTH;
            p1.velocity.y -= collisionDirection.y * d_FORCE_STRENGTH;
        }
    }

    // Update position
    p1.position.x += p1.velocity.x;
    p1.position.y += p1.velocity.y;

    // Bounce off screen edges
    if (p1.position.x >= screenWidth || p1.position.x <= 0) p1.velocity.x *= -1;
    if (p1.position.y >= screenHeight || p1.position.y <= 0) p1.velocity.y *= -1;

    CapSpeed(p1.velocity, d_MAX_SPEED, d_MIN_SPEED);
}


int main() {
    // Screen dimensions
    int screenWidth = 1440;
    int screenHeight = 920;
    InitWindow(screenWidth, screenHeight, "Particle Interaction - CUDA");

    srand(static_cast<unsigned int>(time(0)));

    // Host particles
    Particle* h_particles = new Particle[MAX_PARTICLES];

    // Initialize host particles
    for (int i = 0; i < MAX_PARTICLES; i++) {
        h_particles[i].position = { (float)(rand() % screenWidth), (float)(rand() % screenHeight) };
        h_particles[i].velocity = { (float)(rand() % 5 - 2), (float)(rand() % 5 - 2) };
        h_particles[i].color = { (unsigned char)(rand() % 256), (unsigned char)(rand() % 256), (unsigned char)(rand() % 256), 255 };
    }

    // Initialize constants on the device
    float h_MAX_DISTANCE = 14.0f;
    float h_MIN_DISTANCE = 8.0f;
    float h_FORCE_STRENGTH = 5.0f;
    float h_MIN_COLLISION_DISTANCE = 10.0f;
    float h_MAX_SPEED = 2.5f;
    float h_MIN_SPEED = 0.1f;

    hipMemcpyToSymbol(HIP_SYMBOL(d_MAX_DISTANCE), &h_MAX_DISTANCE, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_MIN_DISTANCE), &h_MIN_DISTANCE, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_FORCE_STRENGTH), &h_FORCE_STRENGTH, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_MIN_COLLISION_DISTANCE), &h_MIN_COLLISION_DISTANCE, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_MAX_SPEED), &h_MAX_SPEED, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_MIN_SPEED), &h_MIN_SPEED, sizeof(float));

    // Device particles
    Particle* d_particles;
    hipMalloc(&d_particles, MAX_PARTICLES * sizeof(Particle));
    hipMemcpy(d_particles, h_particles, MAX_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);

    SetTargetFPS(144);

    while (!WindowShouldClose()) {
        // Launch kernel
        int blocks = (MAX_PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE;
        UpdateParticleInteractions << <blocks, BLOCK_SIZE >> > (d_particles, MAX_PARTICLES, screenWidth, screenHeight);
        hipDeviceSynchronize();

        // Copy updated particles back to host
        hipMemcpy(h_particles, d_particles, MAX_PARTICLES * sizeof(Particle), hipMemcpyDeviceToHost);

        BeginDrawing();
        ClearBackground(BLACK);

        for (int i = 0; i < MAX_PARTICLES; i++) {
            DrawCircleV(h_particles[i].position, 7.0f, h_particles[i].color);
        }

        DrawText(TextFormat("FPS: %i", GetFPS()), 10, 10, 20, WHITE);
        EndDrawing();
    }

    // Cleanup
    delete[] h_particles;
    hipFree(d_particles);
    CloseWindow();

    return 0;
}