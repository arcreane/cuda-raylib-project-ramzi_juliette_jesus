#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <raylib.h>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <vector>

// Constants in device memory
__constant__ int d_screenWidth;
__constant__ int d_screenHeight;
__constant__ float d_FORCE_STRENGTH;
__constant__ float d_MIN_DISTANCE;
__constant__ float d_MAX_DISTANCE;
__constant__ float d_MAX_SPEED;
__constant__ float d_MIN_SPEED;
__constant__ float d_MIN_COLLISION_DISTANCE;
__constant__ float d_radius;

// Constants for display and shape
int h_screenWidth = 1920;
int h_screenHeight = 920;
float h_radius = 7.0f;

// Constants for interaction and collision
float h_FORCE_STRENGTH = 5.0f; // Attraction/repulsion force constant 
float h_MIN_DISTANCE = 2 * h_radius* 2 * h_radius; // Minimum distance for interaction (avoid division by zero)
float h_MAX_DISTANCE = 2.8 * h_radius* 2.8 * h_radius; // Maximum distance for interaction (particles won't affect each other beyond this)
float h_MAX_SPEED = 2.5f; // Maximum speed for particles
float h_MIN_SPEED = 0.1f; // Minimum speed for particles
float h_MIN_COLLISION_DISTANCE = 2.5 * h_radius * 2.5 * h_radius; // Minimum distance for particles to collide and bounce

// Flag to pause
bool pause = 0;

#define MAX_PARTICLES 2000
#define BLOCK_SIZE 256

/////// CPU Performance Notes //////////
// 2000 = 14 FPS                      //
// 1000 = 53 FPS                      //
// 500 = 142 FPS                      //
////////////////////////////////////////
/////// GPU First Version 1 Notes //////
// 5000 = 40 FPS                      //
// 4000 = 50 FPS                      //
// 3000 = 66 FPS                      //
// 2000 = 96 FPS                      //
// 1000 = 142 FPS                     //
// 500 = 144 FPS                      //
////////////////////////////////////////

// Particle struct definition
struct Particle {
    Vector2 position;
    Vector2 velocity;
    Color color;
};

// CUDA Kernel to limit the velocity of a particle to the maximum and minimum speeds
__device__ void CapSpeed(Vector2& velocity, float maxSpeed, float minSpeed) {
    float speed = sqrtf(velocity.x * velocity.x + velocity.y * velocity.y);
    if (speed > maxSpeed) {
        velocity.x = (velocity.x / speed) * maxSpeed;
        velocity.y = (velocity.y / speed) * maxSpeed;
    }
    if (speed < minSpeed && speed > 0) {
        velocity.x = (velocity.x / speed) * minSpeed;
        velocity.y = (velocity.y / speed) * minSpeed;
    }
}

// CUDA Kernel to update the position of the particles and control the collision with the walls
__global__ void UpdateParticlesKernel(Particle* particles, int particleCount) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleCount) return;

    Particle& p = particles[i];
    p.position.x += p.velocity.x;
    p.position.y += p.velocity.y;

    // Cap speed
    CapSpeed(p.velocity, d_MAX_SPEED, d_MIN_SPEED);

    // Bounce off edges
    if (p.position.x >= d_screenWidth - d_radius || p.position.x <= d_radius) {
        p.velocity.x *= -1.0f;
        p.position.x = fminf(fmaxf(p.position.x, d_radius), d_screenWidth - d_radius);
    }
    if (p.position.y >= d_screenHeight - d_radius || p.position.y <= d_radius) {
        p.velocity.y *= -1.0f;
        p.position.y = fminf(fmaxf(p.position.y, d_radius), d_screenHeight - d_radius);
    }
}

// CUDA kernel to handle particle interactions
__global__ void HandleInteractionsKernel(Particle* particles, int particleCount) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleCount) return;

    Particle& p1 = particles[i];
    for (int j = 0; j < particleCount; ++j) {
        if (i == j) continue;

        Particle& p2 = particles[j];
        float dx = p2.position.x - p1.position.x;
        float dy = p2.position.y - p1.position.y;
        float distance = dx * dx + dy * dy;

        if (distance < d_MAX_DISTANCE && distance > d_MIN_DISTANCE) {
            float force = -d_FORCE_STRENGTH / distance;
            Vector2 direction = { dx / distance, dy / distance };
            p1.velocity.x += direction.x * force;
            p1.velocity.y += direction.y * force;
        }

        if (distance < d_MIN_COLLISION_DISTANCE) {
            Vector2 collisionDirection = { dx / distance, dy / distance };
            p1.velocity.x -= collisionDirection.x * d_FORCE_STRENGTH;
            p1.velocity.y -= collisionDirection.y * d_FORCE_STRENGTH;
        }
    }
}

// Function to initialize the particles with random values
void InitializeParticles(std::vector<Particle>& particles) {
    for (Particle& particle : particles) {
        particle.position = { (float)(rand() % h_screenWidth), (float)(rand() % h_screenHeight) };
        particle.velocity = { (float)(rand() % 5 - 2), (float)(rand() % 5 - 2) };
        particle.color = Color{ (unsigned char)(rand() % 256), (unsigned char)(rand() % 256), (unsigned char)(rand() % 256), 255 };
    }
}

// CUDA Kernel to check the Keyboard inputs and define the actions to take
__global__ void CheckKeyBoardInputKernel(Particle* particles, int particleCount, bool keyDown, bool keyUp, bool keyLeft, bool keyRight, bool keySpace, bool* pause, float maxSpeed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    Particle& p = particles[i];
    if (keyDown) {
        p.velocity = { 0.0f, maxSpeed };
    }
    if (keyUp) {
        p.velocity = { 0.0f, -maxSpeed };
    }
    if (keyLeft) {
        p.velocity = { -maxSpeed, 0.0f };
    }
    if (keyRight) {
        p.velocity = { maxSpeed, 0.0f };
    }
    if (keySpace) {
        *pause = !(*pause);
    }
}



int main() {

    // Set up window
    InitWindow(h_screenWidth, h_screenHeight, "Particle Interaction - GPU");

    srand(static_cast<unsigned int>(time(0)));

    // Host particles
    std::vector<Particle> h_particles(MAX_PARTICLES);
    InitializeParticles(h_particles);

    // Device particles
    Particle* d_particles;
    hipMalloc(&d_particles, MAX_PARTICLES * sizeof(Particle));
    hipMemcpy(d_particles, h_particles.data(), MAX_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);

    // Pause state
    bool h_pause = false;  // Host pause state
    bool* d_pause;         // Device pause state
    hipMalloc(&d_pause, sizeof(bool));
    hipMemcpy(d_pause, &h_pause, sizeof(bool), hipMemcpyHostToDevice);

    // Copy constants to device
    hipMemcpyToSymbol(HIP_SYMBOL(d_screenWidth), &h_screenWidth, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_screenHeight), &h_screenHeight, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_FORCE_STRENGTH), &h_FORCE_STRENGTH, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_MIN_DISTANCE), &h_MIN_DISTANCE, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_MAX_DISTANCE), &h_MAX_DISTANCE, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_MAX_SPEED), &h_MAX_SPEED, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_MIN_SPEED), &h_MIN_SPEED, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_MIN_COLLISION_DISTANCE), &h_MIN_COLLISION_DISTANCE, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_radius), &h_radius, sizeof(float));

    SetTargetFPS(144);

    while (!WindowShouldClose()) {

        // Gather keyboard input
        bool keyDown = IsKeyDown(KEY_DOWN);
        bool keyUp = IsKeyDown(KEY_UP);
        bool keyLeft = IsKeyDown(KEY_LEFT);
        bool keyRight = IsKeyDown(KEY_RIGHT);
        bool keySpace = IsKeyPressed(KEY_SPACE);  // Check if space is pressed

        // Update the pause state if SPACE is pressed
        if (keySpace) {
            h_pause = !h_pause;  // Toggle pause state on host
            hipMemcpy(d_pause, &h_pause, sizeof(bool), hipMemcpyHostToDevice);  // Sync pause state to device
        }

        if (!h_pause) {
            int blocks = (MAX_PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE;

            // Update particles on GPU
            UpdateParticlesKernel << <blocks, BLOCK_SIZE >> > (d_particles, MAX_PARTICLES);
            hipDeviceSynchronize();

            // Handle interactions on GPU
            HandleInteractionsKernel << <blocks, BLOCK_SIZE >> > (d_particles, MAX_PARTICLES);
            hipDeviceSynchronize();

            // Update velocities based on keyboard input
            CheckKeyBoardInputKernel << <blocks, BLOCK_SIZE >> > (d_particles, MAX_PARTICLES, keyDown, keyUp, keyLeft, keyRight, keySpace, d_pause, h_MAX_SPEED);
            hipDeviceSynchronize();


            // Copy updated particles back to host
            hipMemcpy(h_particles.data(), d_particles, MAX_PARTICLES * sizeof(Particle), hipMemcpyDeviceToHost);
        }

        BeginDrawing();
        ClearBackground(BLACK);
        for (const Particle& particle : h_particles) {
            DrawCircleV(particle.position, h_radius, particle.color);
        }
        DrawText(TextFormat("FPS: %i", GetFPS()), 10, 10, 20, WHITE);
        EndDrawing();
    }

    hipFree(d_particles);
    hipFree(d_pause);
    CloseWindow();

    return 0;
}
